#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "StopWatch.cpp"
#include <sstream>
#include <string>
#include <fstream>
#include <vector>
#include <math.h>

#include "rapidjson/reader.h"
using namespace rapidjson;


using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


__global__ void square(float *d_out, float *d_in){
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f ;
}
__global__ void cube(float *d_out, float *d_in){
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f * f ;
}

struct Point{
    float x,y;
};

int task_square(){
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    float h_in[ARRAY_SIZE];
    for(int i=0;i < ARRAY_SIZE;i++){
        h_in[i] = float(i);
    }

    float h_out[ARRAY_SIZE];

    float * d_in;
    float * d_out;


    cout << "Running square" << endl;
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);


    hipMemcpy(d_in, h_in, ARRAY_BYTES,hipMemcpyHostToDevice );

    square<<<1, ARRAY_SIZE>>>(d_out, d_in);

    hipMemcpy(h_out, d_out, ARRAY_BYTES,hipMemcpyDeviceToHost );
    for(int i=0;i < ARRAY_SIZE;i++){
        cout << h_out[i] ;
        cout << (((i%4)!=3) ? "\t" : "\n");
    }

// For cube
    cout << "\n" ;
    cout << "Running Cube" << endl;
    for(int i=0;i < ARRAY_SIZE;i++){
        h_in[i] = float(i);
    }
    hipMemcpy(d_in, h_in, ARRAY_BYTES,hipMemcpyHostToDevice );

    cube<<<1, ARRAY_SIZE>>>(d_out, d_in);
    hipMemcpy(h_out, d_out, ARRAY_BYTES,hipMemcpyDeviceToHost );
    for(int i=0;i < ARRAY_SIZE;i++){
        cout << h_out[i] ;
        cout << (((i%4)!=3) ? "\t" : "\n");
    }




    hipFree(d_in);
    hipFree(d_out);

    return 0;
}

int cuda_sample_code(){
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
           c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

int task_polygon(int argc,char *argv){
    cout << "Running Task4 to find shortest path in Polygon Maps" << endl;

    if(argc !=1)
    {
        cout << "Input file not found" << endl ;
        //exit(1);
        return 0;
    }

    string input_file_name = argv;

    cout << "Processing input file : " << input_file_name << endl;
    std::ifstream infile(input_file_name);
    string line;

    std::getline(infile, line);
    std::istringstream iss(line);


    Point start;
    iss >> start.x ;
    iss >> start.y ;

    std::getline(infile, line);
    std::istringstream isse(line);


    Point end;
    isse >> end.x ;
    isse >> end.y ;


    vector<vector<Point>> polygons;

    std::getline(infile, line); // empty line


    vector<Point> temp_point;
    while (std::getline(infile, line))
    {
        std::istringstream iss(line);

        if(line == "")
        {
            polygons.push_back(temp_point) ;
            temp_point.clear();
        }
        else
        {
            Point temp;
            iss >> temp.x ;
            iss >> temp.y ;

            temp_point.push_back(temp);

        }
    }

    cout << "total number of polygons : " << polygons.size() << endl;

    double distance = sqrt( pow((end.x - start.x),2) +  pow(( end.y - start.y),2)) ;

    cout << "Shortest possible distance(Ignoring Obstacles): " << distance << endl ;


    for(int i=0;i< polygons.size();i++){
        temp_point = polygons.at(i);
        for(int j=0;j< temp_point.size();j++){
            Point temp = temp_point.at(j);

            //cout << temp.x << " " << temp.y << endl;
        }

        //cout << "\n" ;
    }


    ofstream vis_graph_file;

    remove( "points.txt" );
    vis_graph_file.open ("points.txt",  ios::out | ios::app);


    vector<Point> vis_graph_points;


    for(int i=0;i< polygons.size();i++){
        temp_point = polygons.at(i);  // Each polygon
        for(int j=0;j< temp_point.size();j++){
            Point temp = temp_point.at(j);

            vis_graph_points.push_back(start);
            vis_graph_points.push_back(temp);

            /*

            for(int k=0 ; k < temp_point.size() ; k++ ){
                Point next_point = temp_point.at(k);
                vis_graph_points.push_back(next_point);

                //vis_graph_file << "\n" ;
            }
*/
            //vis_graph_file << "\n" ;
        }

        //vis_graph_file << "\n" ;
    }



    for(int i=0;i< polygons.size();i++){
        temp_point = polygons.at(i);  // Each polygon
        for(int j=0;j< temp_point.size();j++){
            Point temp = temp_point.at(j);

            vis_graph_points.push_back(temp);
            vis_graph_points.push_back(end);


/*
						for(int k=0 ; k < temp_point.size() ; k++ ){
							Point next_point = temp_point.at(k);
							vis_graph_points.push_back(next_point);

							//vis_graph_file << "\n" ;
						}
*/
            //vis_graph_file << "\n" ;
        }

        //vis_graph_file << "\n" ;
    }


    for(int i=0; i< vis_graph_points.size(); i++){
        Point vis_points = vis_graph_points.at(i);

        vis_graph_file << vis_points.x << " " << vis_points.y << endl;
    }



    vis_graph_file.close();


//	double time = stopWatch.elapsedTime();

    //cout << "Total Execution Time : " << time << endl;




    return 0;
}



struct MyHandler {
    bool Null() { cout << "Null()" << endl; return true; }
    bool Bool(bool b) { cout << "Bool(" << boolalpha << b << ")" << endl; return true; }
    bool Int(int i) { cout << "Int(" << i << ")" << endl; return true; }
    bool Uint(unsigned u) { cout << "Uint(" << u << ")" << endl; return true; }
    bool Int64(int64_t i) { cout << "Int64(" << i << ")" << endl; return true; }
    bool Uint64(uint64_t u) { cout << "Uint64(" << u << ")" << endl; return true; }
    bool Double(double d) { cout << "Double(" << d << ")" << endl; return true; }
    bool RawNumber(const char* str, SizeType length, bool copy) {
        cout << "Number(" << str << ", " << length << ", " << boolalpha << copy << ")" << endl;
        return true;
    }
    bool String(const char* str, SizeType length, bool copy) {
        cout << "String(" << str << ", " << length << ", " << boolalpha << copy << ")" << endl;
        return true;
    }
    bool StartObject() { cout << "StartObject()" << endl; return true; }
    bool Key(const char* str, SizeType length, bool copy) {
        cout << "Key(" << str << ", " << length << ", " << boolalpha << copy << ")" << endl;
        return true;
    }
    bool EndObject(SizeType memberCount) { cout << "EndObject(" << memberCount << ")" << endl; return true; }
    bool StartArray() { cout << "StartArray()" << endl; return true; }
    bool EndArray(SizeType elementCount) { cout << "EndArray(" << elementCount << ")" << endl; return true; }
};

int read_json_data(string file_path)
{

    string input_file_name = file_path;

    cout << "Processing input file : " << input_file_name << endl;
    std::ifstream infile(input_file_name);
    string line;

    if(infile){
        cout << "file not found : "<< file_path << endl;
    }
    std::getline(infile, line);
    std::stringstream iss(line);

    cout << line << iss.str() <<endl;
    const char json[] = " { \"hello\" : \"world\", \"t\" : true , \"f\" : false, \"n\": null, \"i\":123, \"pi\": 3.1416, \"a\":[1, 2, 3, 4] } ";

    MyHandler handler;
    Reader reader;
    StringStream ss(json);
    reader.Parse(ss, handler);

   // StringStream data(iss);
    StringStream ss_data(iss.str().c_str());
    reader.Parse(ss_data, handler);


    return 0;
}
int main()
{
    StopWatch* stopWatch= new StopWatch();
    stopWatch->start();

    //int task_square_status = task_square();

    //int cudaStatus = cuda_sample_code();

    //char* filePath="resources/data/polygons_300.txt";
    //int task_polygon_status = task_polygon(1,filePath);

    string json_file_path="resources/data.json";
    int data_read_status = read_json_data(json_file_path);
    double time = stopWatch->elapsedTime();
    cout << "Total Time" << time;
    return 0;

}
